#include <hip/hip_runtime.h>
#include <stddef.h>

__global__
void scalar_multiplier_kernel(float *A, size_t rows, size_t cols, float c) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < rows * cols) {
    A[idx] *= c;
  }
}

extern "C"
void scalar_multiplier(float *A_h, size_t rows, size_t cols, float c) {
  float *A_d; size_t size = rows * cols * sizeof(float);
  hipMalloc(&A_d, size);
  hipMemcpy(A_d, A_h, size, hipMemcpyHostToDevice);

  int threads = 256;
  int blocks = (rows * cols + threads - 1) / threads;
  scalar_multiplier_kernel<<<blocks, threads>>>(A_d, rows, cols, c);
  hipDeviceSynchronize();

  hipMemcpy(A_h, A_d, size, hipMemcpyDeviceToHost);
  hipFree(A_d);
}
