#include <hip/hip_runtime.h>
#include <stddef.h>

#define CEIL_DIV(x, y) ((x + y - 1) / y)
#define IDX(i, j, ldm) ((i) * ldm + (j))  // ldm = leading dimension (if the 2d array is row major, ldm = cols)
#ifndef SWAP
#define SWAP(x, y) {float *tmp=x; x=y; y=tmp;}
#endif

__global__
void scalar_multiplier_kernel(float *A, size_t rows, size_t cols, float c) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  if (x < rows * cols)
    A[x] *= c;
}

__global__
void mat_add_kernel(float *A, float *B, size_t rows, size_t cols, float dt) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  if (x < rows * cols)
    A[x] += (dt * B[x]);
}

__global__
void diffuse_bad_kernel(float *A, float *B, size_t rows, size_t cols, float a) {
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  int i = blockIdx.y * blockDim.y + threadIdx.y;

  // skip the first and last row and column
  if (1 <= i && i <= rows-2 && 1 <= j && j <= cols-2)
    A[IDX(i, j, cols)] = B[IDX(i, j, cols)] + a * (B[IDX(i-1, j, cols)] + B[IDX(i+1, j, cols)] + B[IDX(i, j-1, cols)] + B[IDX(i, j+1, cols)] - 4 * B[IDX(i, j, cols)]);
}

__global__
void set_bnd_kernel(float *A, size_t rows, size_t cols, int b) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  // skip the corners as they have already been taken care of be the host stub function
  if (1 <= i && i <= rows-2) {
    A[IDX(i,      0, cols)] = b == 2 ? -A[IDX(i,      1, cols)] : A[IDX(i,      1, cols)];
    A[IDX(i, cols-1, cols)] = b == 2 ? -A[IDX(i, cols-2, cols)] : A[IDX(i, cols-2, cols)];
  }
  if (1 <= i && i <= cols-2) {
    A[IDX(     0, i, cols)] = b == 1 ? -A[IDX(     1, i, cols)] : A[IDX(     1, i, cols)];
    A[IDX(rows-1, i, cols)] = b == 1 ? -A[IDX(rows-2, i, cols)] : A[IDX(rows-2, i, cols)];
  }
}

__global__
void diffuse_jacobi_kernel(float *A, float *B, const float *C, size_t rows, size_t cols, float a) {
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  int i = blockIdx.y * blockDim.y + threadIdx.y;

  if (1 <= i && i <= rows-2 && 1 <= j && j <= cols-2)
    A[IDX(i,j,cols)] = (C[IDX(i,j,cols)] + a * (B[IDX(i-1,j,cols)] + B[IDX(i+1,j,cols)] + B[IDX(i,j-1,cols)] + B[IDX(i,j+1,cols)])) / (1.0f + 4.0f * a);
}

__global__
void advect_kernel(float *d, float *d0, float *u, float *v, size_t rows, size_t cols, float dt0) {
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  int i = blockIdx.y * blockDim.y + threadIdx.y;

  if (1 <= i && i <= rows-2 && 1 <= j && j <= cols - 2) {
      float x = i - dt0 * u[IDX(i, j, cols)];
      float y = j - dt0 * v[IDX(i, j, cols)];
      if (x < 0.5f)
        x = 0.5f;
      if (x > rows + 0.5f)
        x = rows + 0.5f;
      int i0 = (int)x;
      int i1 = i0 + 1;
      if (y < 0.5f)
        y = 0.5f;
      if (y > cols + 0.5f)
        y = cols + 0.5f;
      int j0 = (int)y;
      int j1 = j0 + 1;

      float s1 = x - i0;
      float s0 = 1 - s1;
      float t1 = y - j0;
      float t0 = 1 - t1;

      d[IDX(i, j, cols)] = s0 * (t0 * d0[IDX(i0, j0, cols)] + t1 * d0[IDX(i0, j1, cols)]) +
                    s1 * (t0 * d0[IDX(i1, j0, cols)] + t1 * d0[IDX(i1, j1, cols)]);
  }
}

extern "C"
void scalar_multiplier(float *A_h, size_t rows, size_t cols, float c) {
  float *A_d; size_t size = rows * cols * sizeof(float);
  hipMalloc(&A_d, size);
  hipMemcpy(A_d, A_h, size, hipMemcpyHostToDevice);

  int threads = 256;
  int blocks = CEIL_DIV(rows * cols, threads);
  scalar_multiplier_kernel<<<blocks, threads>>>(A_d, rows, cols, c);
  hipDeviceSynchronize();

  hipMemcpy(A_h, A_d, size, hipMemcpyDeviceToHost);
  hipFree(A_d);
}

extern "C"
void mat_add(float *A_h, float *B_h, size_t rows, size_t cols, float dt) {
  float *A_d, *B_d; size_t size = rows * cols * sizeof(float);
  hipMalloc(&A_d, size);
  hipMalloc(&B_d, size);
  hipMemcpy(A_d, A_h, size, hipMemcpyHostToDevice);
  hipMemcpy(B_d, B_h, size, hipMemcpyHostToDevice);

  int threads = 256;
  int blocks = CEIL_DIV(rows * cols, threads);
  mat_add_kernel<<<blocks, threads>>>(A_d, B_d, rows, cols, dt);
  hipDeviceSynchronize();

  hipMemcpy(A_h, A_d, size, hipMemcpyDeviceToHost);
  hipMemcpy(B_h, B_d, size, hipMemcpyDeviceToHost);
  hipFree(A_d);
  hipFree(B_d);
}

extern "C"
void diffuse_bad_host(float *A_h, float *B_h, size_t rows, size_t cols, float a) {
  float *A_d, *B_d; size_t size = rows * cols * sizeof(float);
  hipMalloc(&A_d, size);
  hipMalloc(&B_d, size);
  hipMemcpy(A_d, A_h, size, hipMemcpyHostToDevice);
  hipMemcpy(B_d, B_h, size, hipMemcpyHostToDevice);

  dim3 blockDim(16, 16, 1);
  dim3 gridDim(CEIL_DIV(rows, 16), CEIL_DIV(cols, 16), 1);
  diffuse_bad_kernel<<<gridDim, blockDim>>>(A_d, B_d, rows, cols, a);
  hipDeviceSynchronize();

  hipMemcpy(A_h, A_d, size, hipMemcpyDeviceToHost);
  hipMemcpy(B_h, B_d, size, hipMemcpyDeviceToHost);
  hipFree(A_d);
  hipFree(B_d);
}

extern "C"
void set_bnd_host(float *A_h, size_t rows, size_t cols, int b) {
  // corners
  A_h[IDX(     0,      0, cols)] = 0.5f * (A_h[IDX(     1,      0, cols)] + A_h[IDX(     0,      1, cols)]);
  A_h[IDX(     0, cols-1, cols)] = 0.5f * (A_h[IDX(     1, cols-1, cols)] + A_h[IDX(     0, cols-2, cols)]);
  A_h[IDX(rows-1,      0, cols)] = 0.5f * (A_h[IDX(rows-2,      0, cols)] + A_h[IDX(rows-1,      1, cols)]);
  A_h[IDX(rows-1, cols-1, cols)] = 0.5f * (A_h[IDX(rows-2, cols-1, cols)] + A_h[IDX(rows-1, cols-2, cols)]);

  float *A_d; size_t size = rows * cols * sizeof(float);
  hipMalloc(&A_d, size);
  hipMemcpy(A_d, A_h, size, hipMemcpyHostToDevice);

  int threads = 256;
  int blocks = CEIL_DIV(fmax(rows, cols), threads);
  set_bnd_kernel<<<blocks, threads>>>(A_d, rows, cols, b);
  hipDeviceSynchronize();

  hipMemcpy(A_h, A_d, size, hipMemcpyDeviceToHost);
  hipFree(A_d);
}

extern "C"
void diffuse_jacobi_host(float *A_h, const float *B_h, size_t rows, size_t cols, int b, const float a) {
  float *A_d, *B_d, *C_d; size_t size = rows * cols * sizeof(float);
  hipMalloc(&A_d, size);
  hipMalloc(&B_d, size);
  hipMalloc(&C_d, size);
  hipMemcpy(A_d, A_h, size, hipMemcpyHostToDevice);
  hipMemcpy(B_d, B_h, size, hipMemcpyHostToDevice);

  float *cur   = A_d;      // read buffer (k-th iterate)
  float *next  = C_d;      // write buffer (k+1-th iterate)

  dim3 blockDim(16, 16, 1);
  dim3 gridDim(CEIL_DIV(rows, 16), CEIL_DIV(cols, 16), 1);

  for (int k = 0; k < 20; ++k) {
    diffuse_jacobi_kernel<<<gridDim, blockDim>>>(next, cur, B_d, rows, cols, a);
    set_bnd_kernel<<<CEIL_DIV(fmax(rows, cols), 256), 256>>>(next, rows, cols, b);
    SWAP(cur, next);
  }

  hipDeviceSynchronize();

  // make sure result ends up in the caller's A_h host buffer
  hipMemcpy(A_h, A_d, size, hipMemcpyDeviceToHost);
  hipFree(A_d);
  hipFree(B_d);
  hipFree(C_d);
}

extern "C"
void advect_host(float *d_h, float *d0_h, float *u_h, float *v_h, size_t rows, size_t cols, int b, float dt) {
  float *d_d, *d0_d, *u_d, *v_d; size_t size = rows * cols * sizeof(float);
  hipMalloc(&d_d, size);
  hipMalloc(&d0_d, size);
  hipMalloc(&u_d, size);
  hipMalloc(&v_d, size);
  hipMemcpy(d_d, d_h, size, hipMemcpyHostToDevice);
  hipMemcpy(d0_d, d0_h, size, hipMemcpyHostToDevice);
  hipMemcpy(u_d, u_h, size, hipMemcpyHostToDevice);
  hipMemcpy(v_d, v_h, size, hipMemcpyHostToDevice);

  dim3 blockDim(16, 16, 1);
  dim3 gridDim(CEIL_DIV(rows, 16), CEIL_DIV(cols, 16), 1);

  float dt0 = dt * fmax(rows, cols);
  advect_kernel<<<gridDim, blockDim>>>(d_d, d0_d, u_d, v_d, rows, cols, dt0);
  set_bnd_kernel<<<CEIL_DIV(fmax(rows, cols), 256), 256>>>(d_d, rows, cols, b);
  hipDeviceSynchronize();

  hipMemcpy(d_h, d_d, size, hipMemcpyDeviceToHost);
  hipFree(d_d);
  hipFree(d0_d);
  hipFree(u_d);
  hipFree(v_d);
}
