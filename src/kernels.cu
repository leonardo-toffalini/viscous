#include <hip/hip_runtime.h>
#include <stddef.h>

#define CEIL_DIV(x, y) ((x + y - 1) / y)
#define IDX(i, j, ldm) ((i) * ldm + (j))  // ldm = leading dimension (if the 2d array is row major, ldm = cols)

__global__
void scalar_multiplier_kernel(float *A, size_t rows, size_t cols, float c) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  if (x < rows * cols)
    A[x] *= c;
}

__global__
void mat_add_kernel(float *A, float *B, size_t rows, size_t cols, float dt) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  if (x < rows * cols)
    A[x] += (dt * B[x]);
}

__global__
void diffuse_bad_kernel(float *A, float *B, size_t rows, size_t cols, float a) {
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  int i = blockIdx.y * blockDim.y + threadIdx.y;

  // skip the first and last row and column
  if (1 <= i && i <= rows-2 && 1 <= j && j <= cols-2)
    A[IDX(i, j, cols)] = B[IDX(i, j, cols)] + a * (B[IDX(i-1, j, cols)] + B[IDX(i+1, j, cols)] + B[IDX(i, j-1, cols)] + B[IDX(i, j+1, cols)] - 4 * B[IDX(i, j, cols)]);
}

__global__
void set_bnd_kernel(float *A, size_t rows, size_t cols, int b) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  // skip the corners as they have already been taken care of be the host stub function
  if (1 <= i && i <= rows-2) {
    A[IDX(i,      0, cols)] = b == 2 ? -A[IDX(i,      1, cols)] : A[IDX(i,      1, cols)];
    A[IDX(i, cols-1, cols)] = b == 2 ? -A[IDX(i, cols-2, cols)] : A[IDX(i, cols-2, cols)];
  }
  if (1 <= i && i <= cols-2) {
    A[IDX(     0, i, cols)] = b == 1 ? -A[IDX(     1, i, cols)] : A[IDX(     1, i, cols)];
    A[IDX(rows-1, i, cols)] = b == 1 ? -A[IDX(rows-2, i, cols)] : A[IDX(rows-2, i, cols)];
  }
}

extern "C"
void scalar_multiplier(float *A_h, size_t rows, size_t cols, float c) {
  float *A_d; size_t size = rows * cols * sizeof(float);
  hipMalloc(&A_d, size);
  hipMemcpy(A_d, A_h, size, hipMemcpyHostToDevice);

  int threads = 256;
  int blocks = CEIL_DIV(rows * cols, threads);
  scalar_multiplier_kernel<<<blocks, threads>>>(A_d, rows, cols, c);
  hipDeviceSynchronize();

  hipMemcpy(A_h, A_d, size, hipMemcpyDeviceToHost);
  hipFree(A_d);
}

extern "C"
void mat_add(float *A_h, float *B_h, size_t rows, size_t cols, float dt) {
  float *A_d, *B_d; size_t size = rows * cols * sizeof(float);
  hipMalloc(&A_d, size);
  hipMalloc(&B_d, size);
  hipMemcpy(A_d, A_h, size, hipMemcpyHostToDevice);
  hipMemcpy(B_d, B_h, size, hipMemcpyHostToDevice);

  int threads = 256;
  int blocks = CEIL_DIV(rows * cols, threads);
  mat_add_kernel<<<blocks, threads>>>(A_d, B_d, rows, cols, dt);
  hipDeviceSynchronize();

  hipMemcpy(A_h, A_d, size, hipMemcpyDeviceToHost);
  hipMemcpy(B_h, B_d, size, hipMemcpyDeviceToHost);
  hipFree(A_d);
  hipFree(B_d);
}

extern "C"
void diffuse_bad_host(float *A_h, float *B_h, size_t rows, size_t cols, float a) {
  float *A_d, *B_d; size_t size = rows * cols * sizeof(float);
  hipMalloc(&A_d, size);
  hipMalloc(&B_d, size);
  hipMemcpy(A_d, A_h, size, hipMemcpyHostToDevice);
  hipMemcpy(B_d, B_h, size, hipMemcpyHostToDevice);

  dim3 blockDim(16, 16, 1);
  dim3 gridDim(CEIL_DIV(rows, 16), CEIL_DIV(cols, 16), 1);
  diffuse_bad_kernel<<<gridDim, blockDim>>>(A_d, B_d, rows, cols, a);
  hipDeviceSynchronize();

  hipMemcpy(A_h, A_d, size, hipMemcpyDeviceToHost);
  hipMemcpy(B_h, B_d, size, hipMemcpyDeviceToHost);
  hipFree(A_d);
  hipFree(B_d);
}

extern "C"
void set_bnd_host(float *A_h, size_t rows, size_t cols, int b) {
  // corners
  A_h[IDX(     0,      0, cols)] = 0.5f * (A_h[IDX(     1,      0, cols)] + A_h[IDX(     0,      1, cols)]);
  A_h[IDX(     0, cols-1, cols)] = 0.5f * (A_h[IDX(     1, cols-1, cols)] + A_h[IDX(     0, cols-2, cols)]);
  A_h[IDX(rows-1,      0, cols)] = 0.5f * (A_h[IDX(rows-2,      0, cols)] + A_h[IDX(rows-1,      1, cols)]);
  A_h[IDX(rows-1, cols-1, cols)] = 0.5f * (A_h[IDX(rows-2, cols-1, cols)] + A_h[IDX(rows-1, cols-2, cols)]);

  float *A_d; size_t size = rows * cols * sizeof(float);
  hipMalloc(&A_d, size);
  hipMemcpy(A_d, A_h, size, hipMemcpyHostToDevice);

  int threads = 256;
  int blocks = CEIL_DIV(fmax(rows, cols), threads);
  set_bnd_kernel<<<blocks, threads>>>(A_d, rows, cols, b);
  hipDeviceSynchronize();

  hipMemcpy(A_h, A_d, size, hipMemcpyDeviceToHost);
  hipFree(A_d);
}
